/*
 * SPDX-License-Identifier: GPL-3.0-only
 *
 * Copyright (C) 2012- The GROMACS Authors
 * Copyright (C) 2025- GaoZheng
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, version 3.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program. If not, see <https://www.gnu.org/licenses/>.
 *
 * ---
 *
 * This file is part of a modified version of the GROMACS molecular simulation package.
 * For details on the original project, consult https://www.gromacs.org.
 *
 * To help fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out https://www.gromacs.org.
 */

/*! \file
 *  \brief Define CUDA implementation of nbnxn_gpu_data_mgmt.h
 *
 *  \author Szilard Pall <pall.szilard@gmail.com>
 */
#include "gmxpre.h"

#include <cassert>
#include <cstdarg>
#include <cstdio>
#include <cstdlib>

// TODO We would like to move this down, but the way NbnxmGpu
//      is currently declared means this has to be before gpu_types.h
#include "nbnxm_cuda_types.h"

// TODO Remove this comment when the above order issue is resolved
#include <cub/device/device_scan.cuh>

#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/gpu_utils/device_context.h"
#include "gromacs/gpu_utils/gpu_utils.h"
#include "gromacs/gpu_utils/gpueventsynchronizer.h"
#include "gromacs/gpu_utils/pmalloc.h"
#include "gromacs/hardware/device_information.h"
#include "gromacs/hardware/device_management.h"
#include "gromacs/math/vectypes.h"
#include "gromacs/mdlib/force_flags.h"
#include "gromacs/mdtypes/interaction_const.h"
#include "gromacs/mdtypes/md_enums.h"
#include "gromacs/nbnxm/atomdata.h"
#include "gromacs/nbnxm/gpu_data_mgmt.h"
#include "gromacs/nbnxm/gridset.h"
#include "gromacs/nbnxm/nbnxm.h"
#include "gromacs/nbnxm/nbnxm_gpu.h"
#include "gromacs/nbnxm/nbnxm_gpu_data_mgmt.h"
#include "gromacs/nbnxm/pairlistsets.h"
#include "gromacs/pbcutil/ishift.h"
#include "gromacs/timing/gpu_timing.h"
#include "gromacs/utility/basedefinitions.h"
#include "gromacs/utility/cstringutil.h"
#include "gromacs/utility/fatalerror.h"
#include "gromacs/utility/real.h"
#include "gromacs/utility/smalloc.h"

#include "nbnxm_cuda.h"
/* Required to stop gcc emitting multiple definition warnings as hip/hip_fp16.h, which is included by
 * device_scan.cuh, doesn't undef __WSB_DEPRECATION_MESSAGE and this is later redefined in
 * device_atomic_functions.h used by nbnxm_cuda_types.h. Seen in cuda 10 and 11 with gcc-11. */
#undef __WSB_DEPRECATION_MESSAGE

namespace Nbnxm
{

/* This is a heuristically determined parameter for the Kepler
 * and Maxwell architectures for the minimum size of ci lists by multiplying
 * this constant with the # of multiprocessors on the current device.
 * Since the maximum number of blocks per multiprocessor is 16, the ideal
 * count for small systems is 32 or 48 blocks per multiprocessor. Because
 * there is a bit of fluctuations in the generated block counts, we use
 * a target of 44 instead of the ideal value of 48.
 */

#if GMX_PTX_ARCH <= 700
static const unsigned int gpu_min_ci_balanced_factor = 44;
#else
/* Updated benchmarking on Ampere, Ada, Hopper shows the ideal count is
 * between 61 and 83 depending on chip */
static const unsigned int gpu_min_ci_balanced_factor = 61;
#endif


void gpu_init_platform_specific(NbnxmGpu* /* nb */)
{
    /* set the kernel type for the current GPU */
    /* pick L1 cache configuration */
    cuda_set_cacheconfig();
}

void gpu_free_platform_specific(NbnxmGpu* /* nb */)
{
    // Nothing specific in CUDA
}

int gpu_min_ci_balanced(NbnxmGpu* nb)
{
    return nb != nullptr ? gpu_min_ci_balanced_factor * nb->deviceContext_->deviceInfo().prop.multiProcessorCount
                         : 0;
}

/* Calculate size of working memory required for exclusive sum, part of sorting the neighbour list,
 * by calling exclusive sum with nullptr */
void getExclusiveScanWorkingArraySize(size_t& scan_size, gpu_plist* d_plist, const DeviceStream& deviceStream)
{
    hipcub::DeviceScan::ExclusiveSum(nullptr,
                                  scan_size,
                                  d_plist->sorting.sciHistogram,
                                  d_plist->sorting.sciOffset,
                                  c_sciHistogramSize,
                                  deviceStream.stream());
}

} // namespace Nbnxm
